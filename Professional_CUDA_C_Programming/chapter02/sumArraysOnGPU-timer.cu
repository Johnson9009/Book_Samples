#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "common.h"
#include <sys/time.h>

void checkResult(float *hostRef, float *gpuRef, const int N) {
  double epsilon = 1.0E-8;
  bool match = true;
  for (int i = 0; i < N; ++i) {
    if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
      match = false;
      printf("Arrays do not match!\n");
      printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
      break;
    }
  }
  if (match == true) {
    printf("Arrays match.\n\n");
  }
}

void initialData(float *data, int size) {
  // Generate different seed for random number
  time_t t;
  srand((unsigned int)time(&t));

  for (int i = 0; i < size; ++i) {
    data[i] = (float)(rand() & 0xFF) / 10.0f;
  }
  return;
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
  for (int i = 0; i < N; ++i) {
    C[i] = A[i] + B[i];
  }
  return;
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) C[i] = A[i] + B[i];
  return;
}

double cpuSecond(void) {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (double(tv.tv_sec) + double(tv.tv_usec) * 1.E-6);
}

int main(int argc, char *argv[]) {
  printf("%s Starting...\n", argv[0]);
  // set up device
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, 0));
  printf("Using Device %d: %s\n", 0, deviceProp.name);
  hipSetDevice(0);
  // set up data size of vectors
  int nElem = 1 << 24;
  printf("Vector size %d\n", nElem);
  // malloc host memory
  size_t nBytes = nElem * sizeof(float);
  float *h_A, *h_B, *hostRef, *gpuRef;
  h_A = (float *)malloc(nBytes);
  h_B = (float *)malloc(nBytes);
  hostRef = (float *)malloc(nBytes);
  gpuRef = (float *)malloc(nBytes);

  // initialize data at host side
  initialData(h_A, nElem);
  initialData(h_B, nElem);

  memset(hostRef, 0, nBytes);
  memset(gpuRef, 0, nBytes);

  // malloc device global memory
  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, nBytes);
  hipMalloc(&d_B, nBytes);
  hipMalloc(&d_C, nBytes);

  // transfer data from host to device
  hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
  // invoke kernel at host side
  dim3 block(64);
  dim3 grid((nElem + block.x - 1) / block.x);

  double iStart = cpuSecond();
  sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
  CHECK(hipDeviceSynchronize());
  double iElaps = cpuSecond() - iStart;
  printf("sumArraysOnGPU <<<%d, %d>>> elapsed %f sec.\n", grid.x, block.x, iElaps);

  // copy kernel result back to host side
  hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

  // add vector at host side for result checks
  iStart = cpuSecond();
  sumArraysOnHost(h_A, h_B, hostRef, nElem);
  iElaps = cpuSecond() - iStart;
  printf("sumArraysOnHost elapsed %f sec.\n", iElaps);

  // check device results
  checkResult(hostRef, gpuRef, nElem);

  // free device global memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // free host memory
  free(h_A);
  free(h_B);
  free(hostRef);
  free(gpuRef);

  return 0;
}
